#include "hip/hip_runtime.h"
#include "common.cuh"

void customAssert(bool condition, const std::string& errorMessage)
{
    if (!condition)
    {
        std::cerr << "Custom assertion failed: " << errorMessage << std::endl;
        assert(condition);
    }
}

void CudaDevice()
{
    hipDeviceProp_t deviceProp;
    int deviceID = 0;

    THROW_IF_CUDA_ERROR(hipSetDevice(deviceID));
    THROW_IF_CUDA_ERROR(hipGetDeviceProperties(&deviceProp, deviceID));
    printf("GPU Device %d: %s (compute capability %d.%d)\n\n", deviceID,
           deviceProp.name, deviceProp.major, deviceProp.minor);
}

float calculate_mean(const float array[], int size)
{
    float sum = 0.0;
    for (int i = 0; i < size; ++i)
    {
        sum += array[i];
    }
    return sum / size;
}

float calculate_standard_deviation(const float array[], int size)
{
    float mean = calculate_mean(array, size);
    float sum_squared_diff = 0.0;

    for (int i = 0; i < size; ++i)
    {
        float diff = array[i] - mean;
        sum_squared_diff += diff * diff;
    }

    float variance = sum_squared_diff / size;
    return std::sqrt(variance);
}

float find_best_average(const float array[], int array_size, int num_elements)
{
    if (num_elements <= 0 || num_elements > array_size)
    {
        std::cerr << "Invalid number of elements." << std::endl;
        return 0.0;
    }

    float max_average = 0.0;

    for (int i = 0; i <= array_size - num_elements; ++i)
    {
        float sum = 0.0;
        for (int j = i; j < i + num_elements; ++j)
        {
            sum += array[j];
        }
        float average = sum / num_elements;
        max_average = std::max(max_average, average);
    }

    return max_average;
}

float find_min_average(const float array[], int array_size, int num_elements)
{
    if (num_elements <= 0 || num_elements > array_size)
    {
        std::cerr << "Invalid number of elements." << std::endl;
        return 0.0;
    }

    float min_average = std::numeric_limits<float>::max();

    for (int i = 0; i <= array_size - num_elements; ++i)
    {
        float sum = 0.0;
        for (int j = i; j < i + num_elements; ++j)
        {
            sum += array[j];
        }
        float average = sum / num_elements;
        min_average = std::min(min_average, average);
    }

    return min_average;
}

template <typename T>
bool check_result(T* input1, T* input2, int size)
{
    bool chk = true;
    for (int i = 0; i < size; i++)
    {
        if (input1[i] != input2[i])
        {
            std::cout << "Error in index: " << i << " -> " << input1[i] << " - "
                      << input2[i] << " ";
            chk = false;
            break;
        }
    }

    // if (chk)
    //     std::cout << "All correct." << std::endl;

    return chk;
}

template bool check_result<unsigned long long>(unsigned long long* input1,
                                               unsigned long long* input2,
                                               int size);