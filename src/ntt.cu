#include "hip/hip_runtime.h"
// (C) Ulvetanna Inc.
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan
// Paper: https://eprint.iacr.org/2023/1410

#include "ntt.cuh"

#define CC_89 // for RTX 4090
// TODO: All Kernel Initialization will be updated with respect to GPUs. (A100, RTX4090, RTX3060Ti)

__device__ void CooleyTukeyUnit(Data& U, Data& V, Root& root, Modulus& modulus)
{
    Data u_ = U;
    Data v_ = VALUE_GPU::mult(V, root, modulus);

    U = VALUE_GPU::add(u_, v_, modulus);
    V = VALUE_GPU::sub(u_, v_, modulus);
}

__device__ void GentlemanSandeUnit(Data& U, Data& V, Root& root,
                                   Modulus& modulus)
{
    Data u_ = U;
    Data v_ = V;

    U = VALUE_GPU::add(u_, v_, modulus);

    v_ = VALUE_GPU::sub(u_, v_, modulus);
    V = VALUE_GPU::mult(v_, root, modulus);
}

__global__ void ForwardCore(Data* polynomial, Root* root_of_unity_table,
                            Modulus modulus, int shared_index, int logm,
                            int outer_iteration_count, int N_power,
                            bool zero_padding, bool not_last_kernel,
                            bool reduction_poly_check)
{
    const int idx_x = threadIdx.x;
    const int idx_y = threadIdx.y;
    const int block_x = blockIdx.x;
    const int block_y = blockIdx.y;
    const int block_z = blockIdx.z;

    extern __shared__ Data shared_memory[];

    int t_2 = N_power - logm - 1;
    location_t offset = 1 << (N_power - logm - 1);
    int t_ = shared_index;
    location_t m = (location_t)1 << logm;

    location_t global_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) +
        (location_t)(blockDim.x * block_x) +
        (location_t)(2 * block_y * offset) + (location_t)(block_z << N_power);
    location_t omega_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) +
        (location_t)(blockDim.x * block_x) + (location_t)(block_y * offset);
    location_t shared_addresss = (idx_x + (idx_y * blockDim.x));

    // Load data from global & store to shared
    shared_memory[shared_addresss] = polynomial[global_addresss];
    shared_memory[shared_addresss + (blockDim.x * blockDim.y)] =
        polynomial[global_addresss + offset];

    int t = 1 << t_;
    int in_shared_address = ((shared_addresss >> t_) << t_) + shared_addresss;
    location_t current_root_index;
    if (not_last_kernel)
    {
#pragma unroll
        for (int lp = 0; lp < outer_iteration_count; lp++)
        {
            __syncthreads();
            if (reduction_poly_check)
            {  // X_N_minus
                current_root_index = (omega_addresss >> t_2);
            }
            else
            {  // X_N_plus
                current_root_index = m + (omega_addresss >> t_2);
            }
            CooleyTukeyUnit(shared_memory[in_shared_address],
                            shared_memory[in_shared_address + t],
                            root_of_unity_table[current_root_index], modulus);

            t = t >> 1;
            t_2 -= 1;
            t_ -= 1;
            m <<= 1;

            in_shared_address =
                ((shared_addresss >> t_) << t_) + shared_addresss;
            //__syncthreads();
        }
        __syncthreads();
    }
    else
    {
#pragma unroll
        for (int lp = 0; lp < (shared_index - 5); lp++) // 4 for 512 thread
        {
            __syncthreads();
            if (reduction_poly_check)
            {  // X_N_minus
                current_root_index = (omega_addresss >> t_2);
            }
            else
            {  // X_N_plus
                current_root_index = m + (omega_addresss >> t_2);
            }

            CooleyTukeyUnit(shared_memory[in_shared_address],
                            shared_memory[in_shared_address + t],
                            root_of_unity_table[current_root_index], modulus);

            t = t >> 1;
            t_2 -= 1;
            t_ -= 1;
            m <<= 1;

            in_shared_address =
                ((shared_addresss >> t_) << t_) + shared_addresss;
            //__syncthreads();
        }
        __syncthreads();

#pragma unroll
        for (int lp = 0; lp < 6; lp++)
        {
            if (reduction_poly_check)
            {  // X_N_minus
                current_root_index = (omega_addresss >> t_2);
            }
            else
            {  // X_N_plus
                current_root_index = m + (omega_addresss >> t_2);
            }
            CooleyTukeyUnit(shared_memory[in_shared_address],
                            shared_memory[in_shared_address + t],
                            root_of_unity_table[current_root_index], modulus);

            t = t >> 1;
            t_2 -= 1;
            t_ -= 1;
            m <<= 1;

            in_shared_address =
                ((shared_addresss >> t_) << t_) + shared_addresss;
        }
        __syncthreads();
    }

    polynomial[global_addresss] = shared_memory[shared_addresss];
    polynomial[global_addresss + offset] =
        shared_memory[shared_addresss + (blockDim.x * blockDim.y)];
}



__global__ void ForwardCore_(Data* polynomial, Root* root_of_unity_table,
                            Modulus modulus, int shared_index, int logm,
                            int outer_iteration_count, int N_power,
                            bool zero_padding, bool not_last_kernel,
                            bool reduction_poly_check)
{
    const int idx_x = threadIdx.x;
    const int idx_y = threadIdx.y;
    const int block_x = blockIdx.x;
    const int block_y = blockIdx.y;
    const int block_z = blockIdx.z;

    extern __shared__ Data shared_memory[];

    int t_2 = N_power - logm - 1;
    location_t offset = 1 << (N_power - logm - 1);
    int t_ = shared_index;
    location_t m = (location_t)1 << logm;

    location_t global_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) +
        (location_t)(blockDim.x * block_y) +
        (location_t)(2 * block_x * offset) + (location_t)(block_z << N_power);
    location_t omega_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) +
        (location_t)(blockDim.x * block_y) + (location_t)(block_x * offset);
    location_t shared_addresss = (idx_x + (idx_y * blockDim.x));

    // Load data from global & store to shared
    shared_memory[shared_addresss] = polynomial[global_addresss];
    shared_memory[shared_addresss + (blockDim.x * blockDim.y)] =
        polynomial[global_addresss + offset];

    int t = 1 << t_;
    int in_shared_address = ((shared_addresss >> t_) << t_) + shared_addresss;
    location_t current_root_index;
    if (not_last_kernel)
    {
#pragma unroll
        for (int lp = 0; lp < outer_iteration_count; lp++)
        {
            __syncthreads();
            if (reduction_poly_check)
            {  // X_N_minus
                current_root_index = (omega_addresss >> t_2);
            }
            else
            {  // X_N_plus
                current_root_index = m + (omega_addresss >> t_2);
            }
            CooleyTukeyUnit(shared_memory[in_shared_address],
                            shared_memory[in_shared_address + t],
                            root_of_unity_table[current_root_index], modulus);

            t = t >> 1;
            t_2 -= 1;
            t_ -= 1;
            m <<= 1;

            in_shared_address =
                ((shared_addresss >> t_) << t_) + shared_addresss;
            //__syncthreads();
        }
        __syncthreads();
    }
    else
    {
#pragma unroll
        for (int lp = 0; lp < (shared_index - 5); lp++)
        {
            __syncthreads();
            if (reduction_poly_check)
            {  // X_N_minus
                current_root_index = (omega_addresss >> t_2);
            }
            else
            {  // X_N_plus
                current_root_index = m + (omega_addresss >> t_2);
            }

            CooleyTukeyUnit(shared_memory[in_shared_address],
                            shared_memory[in_shared_address + t],
                            root_of_unity_table[current_root_index], modulus);

            t = t >> 1;
            t_2 -= 1;
            t_ -= 1;
            m <<= 1;

            in_shared_address =
                ((shared_addresss >> t_) << t_) + shared_addresss;
            //__syncthreads();
        }
        __syncthreads();

#pragma unroll
        for (int lp = 0; lp < 6; lp++)
        {
            if (reduction_poly_check)
            {  // X_N_minus
                current_root_index = (omega_addresss >> t_2);
            }
            else
            {  // X_N_plus
                current_root_index = m + (omega_addresss >> t_2);
            }
            CooleyTukeyUnit(shared_memory[in_shared_address],
                            shared_memory[in_shared_address + t],
                            root_of_unity_table[current_root_index], modulus);

            t = t >> 1;
            t_2 -= 1;
            t_ -= 1;
            m <<= 1;

            in_shared_address =
                ((shared_addresss >> t_) << t_) + shared_addresss;
        }
        __syncthreads();
    }

    polynomial[global_addresss] = shared_memory[shared_addresss];
    polynomial[global_addresss + offset] =
        shared_memory[shared_addresss + (blockDim.x * blockDim.y)];
}

__global__ void InverseCore(Data* polynomial, Root* inverse_root_of_unity_table,
                            Modulus modulus, int shared_index, int logm, int k,
                            int outer_iteration_count, int N_power,
                            Ninverse n_inverse, bool last_kernel,
                            bool reduction_poly_check)
{
    const int idx_x = threadIdx.x;
    const int idx_y = threadIdx.y;
    const int block_x = blockIdx.x;
    const int block_y = blockIdx.y;
    const int block_z = blockIdx.z;

    extern __shared__ Data shared_memory[];

    int t_2 = N_power - logm - 1;
    location_t offset = 1 << (N_power - k - 1);
    //int t_ = 9 - outer_iteration_count;
    int t_ = (shared_index + 1) - outer_iteration_count;
    int loops = outer_iteration_count;
    location_t m = (location_t)1 << logm;

    location_t global_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) +
        (location_t)(blockDim.x * block_x) +
        (location_t)(2 * block_y * offset) + (location_t)(block_z << N_power);

    location_t omega_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) +
        (location_t)(blockDim.x * block_x) + (location_t)(block_y * offset);
    location_t shared_addresss = (idx_x + (idx_y * blockDim.x));

    shared_memory[shared_addresss] = polynomial[global_addresss];
    shared_memory[shared_addresss + (blockDim.x * blockDim.y)] =
        polynomial[global_addresss + offset];

    int t = 1 << t_;
    int in_shared_address = ((shared_addresss >> t_) << t_) + shared_addresss;
    location_t current_root_index;
#pragma unroll
    for (int lp = 0; lp < loops; lp++)
    {
        __syncthreads();
        if (reduction_poly_check)
        {  // X_N_minus
            current_root_index = (omega_addresss >> t_2);
        }
        else
        {  // X_N_plus
            current_root_index = m + (omega_addresss >> t_2);
        }

        GentlemanSandeUnit(shared_memory[in_shared_address],
                           shared_memory[in_shared_address + t],
                           inverse_root_of_unity_table[current_root_index],
                           modulus);

        t = t << 1;
        t_2 += 1;
        t_ += 1;
        m >>= 1;

        in_shared_address = ((shared_addresss >> t_) << t_) + shared_addresss;
    }
    __syncthreads();

    if (last_kernel)
    {
        polynomial[global_addresss] =
            VALUE_GPU::mult(shared_memory[shared_addresss], n_inverse, modulus);
        polynomial[global_addresss + offset] = VALUE_GPU::mult(
            shared_memory[shared_addresss + (blockDim.x * blockDim.y)],
            n_inverse, modulus);
    }
    else
    {
        polynomial[global_addresss] = shared_memory[shared_addresss];
        polynomial[global_addresss + offset] =
            shared_memory[shared_addresss + (blockDim.x * blockDim.y)];
    }
}

__global__ void InverseCore_(Data* polynomial, Root* inverse_root_of_unity_table,
                            Modulus modulus, int shared_index, int logm, int k,
                            int outer_iteration_count, int N_power,
                            Ninverse n_inverse, bool last_kernel,
                            bool reduction_poly_check)
{
    const int idx_x = threadIdx.x;
    const int idx_y = threadIdx.y;
    const int block_x = blockIdx.x;
    const int block_y = blockIdx.y;
    const int block_z = blockIdx.z;

    extern __shared__ Data shared_memory[];

    int t_2 = N_power - logm - 1;
    location_t offset = 1 << (N_power - k - 1);
    //int t_ = 9 - outer_iteration_count;
    int t_ = (shared_index + 1) - outer_iteration_count;
    int loops = outer_iteration_count;
    location_t m = (location_t)1 << logm;

    location_t global_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) +
        (location_t)(blockDim.x * block_y) +
        (location_t)(2 * block_x * offset) + (location_t)(block_z << N_power);

    location_t omega_addresss =
        idx_x +
        (location_t)(idx_y * (offset / (1 << (outer_iteration_count - 1)))) +
        (location_t)(blockDim.x * block_y) + (location_t)(block_x * offset);
    location_t shared_addresss = (idx_x + (idx_y * blockDim.x));

    shared_memory[shared_addresss] = polynomial[global_addresss];
    shared_memory[shared_addresss + (blockDim.x * blockDim.y)] =
        polynomial[global_addresss + offset];

    int t = 1 << t_;
    int in_shared_address = ((shared_addresss >> t_) << t_) + shared_addresss;
    location_t current_root_index;
#pragma unroll
    for (int lp = 0; lp < loops; lp++)
    {
        __syncthreads();
        if (reduction_poly_check)
        {  // X_N_minus
            current_root_index = (omega_addresss >> t_2);
        }
        else
        {  // X_N_plus
            current_root_index = m + (omega_addresss >> t_2);
        }

        GentlemanSandeUnit(shared_memory[in_shared_address],
                           shared_memory[in_shared_address + t],
                           inverse_root_of_unity_table[current_root_index],
                           modulus);

        t = t << 1;
        t_2 += 1;
        t_ += 1;
        m >>= 1;

        in_shared_address = ((shared_addresss >> t_) << t_) + shared_addresss;
    }
    __syncthreads();

    if (last_kernel)
    {
        polynomial[global_addresss] =
            VALUE_GPU::mult(shared_memory[shared_addresss], n_inverse, modulus);
        polynomial[global_addresss + offset] = VALUE_GPU::mult(
            shared_memory[shared_addresss + (blockDim.x * blockDim.y)],
            n_inverse, modulus);
    }
    else
    {
        polynomial[global_addresss] = shared_memory[shared_addresss];
        polynomial[global_addresss + offset] =
            shared_memory[shared_addresss + (blockDim.x * blockDim.y)];
    }
}

__host__ void GPU_NTT(Data* device_inout, Root* root_of_unity_table,
                      Modulus modulus, ntt_configuration cfg, int batch_size)
{
    switch (cfg.ntt_type)
    {
        case FORWARD:
            switch (cfg.n_power)
            {
                case 12:
                    ForwardCore<<<dim3(8, 1, batch_size), dim3(64, 4),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 3,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 8, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 3, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 13:
                    ForwardCore<<<dim3(16, 1, batch_size), dim3(32, 8),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 4,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 16, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 4, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 14:
                    ForwardCore<<<dim3(32, 1, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 5,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 32, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 5, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 15:
                    ForwardCore<<<dim3(64, 1, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 6,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 64, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 6, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 16:
                    ForwardCore<<<dim3(128, 1, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 7,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 128, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 7, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 17:
                    ForwardCore<<<dim3(256, 1, batch_size), dim3(32, 8),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 4,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(16, 16, batch_size), dim3(32, 8),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 4, 4,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 256, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 8, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 18:
                    ForwardCore<<<dim3(512, 1, batch_size), dim3(32, 8),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 4,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(32, 16, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 4, 5,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 512, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 9, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 19:
                    ForwardCore<<<dim3(1024, 1, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 5,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(32, 32, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 5, 5,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 1024, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 10, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 20:
                    ForwardCore<<<dim3(2048, 1, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 5,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(64, 32, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 5, 6,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 2048, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 11, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 21:
                    ForwardCore<<<dim3(4096, 1, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 6,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(64, 64, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 6, 6,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 4096, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 12, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 22:
                    ForwardCore<<<dim3(8192, 1, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 6,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(128, 64, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 6, 7,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 8192, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 13, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());

                    break;
                case 23:
                    ForwardCore<<<dim3(16384, 1, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 7,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(128, 128, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 7, 7,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 16384, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 14, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 24:
                    ForwardCore<<<dim3(16384, 1, batch_size), dim3(8, 64),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 0, 7,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(128, 128, batch_size), dim3(8, 64),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 7, 7,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(1, 16384, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 14, 10,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 25:
                    ForwardCore<<<dim3(32768, 1, batch_size), dim3(8, 64),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 0, 7,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(256, 128, batch_size), dim3(4, 128),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 7, 8,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    ForwardCore_<<<dim3(32768, 1, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 15, 10,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 26:
                    ForwardCore<<<dim3(65536, 1, batch_size), dim3(4, 128),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 0, 8,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(256, 256, batch_size), dim3(4, 128),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 8, 8,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    ForwardCore_<<<dim3(65536, 1, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 16, 10,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 27:
#ifndef CC_89
                    ForwardCore<<<dim3(262144, 1, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 5,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(8192, 32, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 5, 6,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(128, 2048, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 11, 7,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    ForwardCore_<<<dim3(262144, 1, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 18, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
#else
                    ForwardCore<<<dim3(131072, 1, batch_size), dim3(4, 128),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 0, 8,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(512, 256, batch_size), dim3(2, 256),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 8, 9,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    ForwardCore_<<<dim3(131072, 1, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 17, 10,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
#endif
                    break;
                case 28:
#ifndef CC_89
                    ForwardCore<<<dim3(524288, 1, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 0, 6,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(8192, 64, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 6, 6,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(128, 4096, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 12, 7,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    ForwardCore_<<<dim3(524288, 1, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 19, 9,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
#else
                    ForwardCore<<<dim3(262144, 1, batch_size), dim3(2, 256),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 0, 9,
                        cfg.n_power, cfg.zero_padding, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ForwardCore<<<dim3(512, 512, batch_size), dim3(2, 256),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 9, 9,
                        cfg.n_power, false, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    ForwardCore_<<<dim3(262144, 1, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 18, 10,
                        cfg.n_power, false, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
#endif
                    break;

                default:
                    break;
            }
            break;
        case INVERSE:
            switch (cfg.n_power)
            {
                case 12:
                    InverseCore<<<dim3(1, 8, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 11, 3, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(8, 1, batch_size), dim3(64, 4),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 2, 0, 3,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 13:
                    InverseCore<<<dim3(1, 16, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 12, 4, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(16, 1, batch_size), dim3(32, 8),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 3, 0, 4,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 14:
                    InverseCore<<<dim3(1, 32, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 13, 5, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(32, 1, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 4, 0, 5,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 15:
                    InverseCore<<<dim3(1, 64, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 14, 6, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(64, 1, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 5, 0, 6,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 16:
                    InverseCore<<<dim3(1, 128, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 15, 7, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(128, 1, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 6, 0, 7,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 17:
                    InverseCore<<<dim3(1, 256, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 16, 8, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(16, 16, batch_size), dim3(32, 8),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 7, 4, 4,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(256, 1, batch_size), dim3(32, 8),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 3, 0, 4,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 18:
                    InverseCore<<<dim3(1, 512, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 17, 9, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(32, 16, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 8, 4, 5,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(512, 1, batch_size), dim3(32, 8),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 3, 0, 4,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 19:
                    InverseCore<<<dim3(1, 1024, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 18, 10, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(32, 32, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 9, 5, 5,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(1024, 1, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 4, 0, 5,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 20:
                    InverseCore<<<dim3(1, 2048, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 19, 11, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(64, 32, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 10, 5, 6,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(2048, 1, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 4, 0, 5,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 21:  //
                    InverseCore<<<dim3(1, 4096, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 20, 12, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(64, 64, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 11, 6, 6,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(4096, 1, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 5, 0, 6,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 22:
                    InverseCore<<<dim3(1, 8192, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 21, 13, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(128, 64, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 12, 6, 7,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(8192, 1, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 5, 0, 6,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 23:
                    InverseCore<<<dim3(1, 16384, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 22, 14, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(128, 128, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 13, 7, 7,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(16384, 1, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 6, 0, 7,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 24:
                    InverseCore<<<dim3(1, 16384, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 23, 14, 10,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(128, 128, batch_size), dim3(8, 64),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 13, 7, 7,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(16384, 1, batch_size), dim3(8, 64),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 6, 0, 7,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 25:
                    InverseCore_<<<dim3(32768, 1, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 24, 15, 10,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    InverseCore<<<dim3(256, 128, batch_size), dim3(4, 128),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 14, 7, 8,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(32768, 1, batch_size), dim3(8, 64),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 6, 0, 7,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    break;
                case 26:
                    InverseCore_<<<dim3(65536, 1, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 25, 16, 10,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    InverseCore<<<dim3(256, 256, batch_size), dim3(4, 128),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 15, 8, 8,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(65536, 1, batch_size), dim3(4, 128),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 7, 0, 8,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());    
                    break;
                case 27:
#ifndef CC_89
                    InverseCore_<<<dim3(262144, 1, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 26, 18, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    InverseCore<<<dim3(128, 2048, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 17, 11, 7,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(8192, 32, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 10, 5, 6,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(262144, 1, batch_size), dim3(16, 16),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 4, 0, 5,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
#else
                    InverseCore_<<<dim3(131072, 1, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 26, 17, 10,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    InverseCore<<<dim3(512, 256, batch_size), dim3(2, 256),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 16, 8, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(131072, 1, batch_size), dim3(4, 128),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 7, 0, 8,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
#endif
                    break;
                case 28:
#ifndef CC_89
                    InverseCore_<<<dim3(524288, 1, batch_size), dim3(256, 1),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 27, 19, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    InverseCore<<<dim3(128, 4096, batch_size), dim3(4, 64),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 18, 12, 7,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(8192, 64, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 11, 6, 6,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(524288, 1, batch_size), dim3(8, 32),
                                  512 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 8, 5, 0, 6,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
#else
                    InverseCore_<<<dim3(262144, 1, batch_size), dim3(512, 1),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 27, 18, 10,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    ///////////////////////////////////////////////////////////
                    InverseCore<<<dim3(512, 512, batch_size), dim3(2, 256),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 17, 9, 9,
                        cfg.n_power, cfg.mod_inverse, false,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
                    InverseCore<<<dim3(262144, 1, batch_size), dim3(2, 256),
                                  1024 * sizeof(Data), cfg.stream>>>(
                        device_inout, root_of_unity_table, modulus, 9, 8, 0, 9,
                        cfg.n_power, cfg.mod_inverse, true,
                        (cfg.reduction_poly == ReductionPolynomial::X_N_minus));
                    THROW_IF_CUDA_ERROR(hipGetLastError());
#endif
                    break;

                default:
                    break;
            }
            break;

        default:
            break;
    }
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void GPU_ACTIVITY(unsigned long long* output,
                             unsigned long long fix_num)
{
    int idx = blockIdx.x + blockDim.x + threadIdx.x;

    output[idx] = fix_num;
}

__host__ void GPU_ACTIVITY_HOST(unsigned long long* output,
                                unsigned long long fix_num)
{
    GPU_ACTIVITY<<<64, 512>>>(output, fix_num);
}


__global__ void GPU_ACTIVITY2(unsigned long long* input1, unsigned long long* input2)
{
    int idx = blockIdx.x + blockDim.x + threadIdx.x;

    input1[idx] = input1[idx] + input2[idx];
}

__host__ void GPU_ACTIVITY2_HOST(unsigned long long* input1, unsigned long long* input2,
                                unsigned size)
{
    GPU_ACTIVITY2<<<(size >> 8), 256>>>(input1, input2);
}