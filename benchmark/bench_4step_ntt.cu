// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include <cstdlib>
#include <random>

#include "ntt_4step.cuh"
#include "ntt_4step_cpu.cuh"

#define DEFAULT_MODULUS

using namespace std;
using namespace gpuntt;

int LOGN;
int BATCH;

int main(int argc, char* argv[])
{
    CudaDevice();

    if (argc < 3)
    {
        LOGN = 12;
        BATCH = 1;
    }
    else
    {
        LOGN = atoi(argv[1]);
        BATCH = atoi(argv[2]);

        if ((LOGN < 12) || (24 < LOGN))
        {
            throw std::runtime_error("LOGN should be in range 12 to 24.");
        }
    }

    ModularReductionType modular_reduction_type = ModularReductionType::BARRET;

    // Current 4step NTT implementation only works for
    // ReductionPolynomial::X_N_minus!
    NTTParameters4Step<Data64> parameters(LOGN, modular_reduction_type,
                                          ReductionPolynomial::X_N_minus);

    int N = parameters.n;

    const int test_count = 100;
    const int bestof = 25;
    float time_measurements[test_count];
    for (int loop = 0; loop < test_count; loop++)
    {
        std::random_device rd;
        std::mt19937 gen(rd());
        unsigned long long minNumber = (unsigned long long) 1 << 40;
        unsigned long long maxNumber = ((unsigned long long) 1 << 40) - 1;
        std::uniform_int_distribution<unsigned long long> dis(minNumber,
                                                              maxNumber);
        unsigned long long number = dis(gen);

        std::uniform_int_distribution<unsigned long long> dis2(0, number);

        Modulus64 modulus(number);

        // Random data generation for polynomials
        vector<vector<Data64>> input1(BATCH);
        for (int j = 0; j < BATCH; j++)
        {
            for (int i = 0; i < N; i++)
            {
                input1[j].push_back(dis2(gen));
            }
        }

        vector<Root64> forward_root_table1;

        for (int i = 0; i < (parameters.n1 >> 1); i++)
        {
            forward_root_table1.push_back(dis2(gen));
        }
        Ninverse64 n_inv = dis2(gen);

        vector<Root64> forward_root_table2;
        for (int i = 0; i < (parameters.n2 >> 1); i++)
        {
            forward_root_table2.push_back(dis2(gen));
        }

        vector<Root64> W_root_table;
        for (int i = 0; i < parameters.n; i++)
        {
            W_root_table.push_back(dis2(gen));
        }

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        Data64* Input_Datas;

        GPUNTT_CUDA_CHECK(hipMalloc(&Input_Datas, BATCH * N * sizeof(Data64)));

        for (int j = 0; j < BATCH; j++)
        {
            GPUNTT_CUDA_CHECK(hipMemcpy(Input_Datas + (N * j),
                                         input1[j].data(), N * sizeof(Data64),
                                         hipMemcpyHostToDevice));
        }

        Data64* Output_Datas;

        GPUNTT_CUDA_CHECK(
            hipMalloc(&Output_Datas, BATCH * N * sizeof(Data64)));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        Root64* Forward_Omega_Table1_Device;
        GPUNTT_CUDA_CHECK(hipMalloc(&Forward_Omega_Table1_Device,
                                     (parameters.n1 >> 1) * sizeof(Root64)));
        GPUNTT_CUDA_CHECK(hipMemcpy(
            Forward_Omega_Table1_Device, forward_root_table1.data(),
            (parameters.n1 >> 1) * sizeof(Root64), hipMemcpyHostToDevice));

        Root64* Forward_Omega_Table2_Device;
        GPUNTT_CUDA_CHECK(hipMalloc(&Forward_Omega_Table2_Device,
                                     (parameters.n2 >> 1) * sizeof(Root64)));
        GPUNTT_CUDA_CHECK(hipMemcpy(
            Forward_Omega_Table2_Device, forward_root_table2.data(),
            (parameters.n2 >> 1) * sizeof(Root64), hipMemcpyHostToDevice));

        Root64* W_Table_Device;
        GPUNTT_CUDA_CHECK(
            hipMalloc(&W_Table_Device, parameters.n * sizeof(Root64)));
        GPUNTT_CUDA_CHECK(hipMemcpy(W_Table_Device, W_root_table.data(),
                                     (parameters.n >> 1) * sizeof(Root64),
                                     hipMemcpyHostToDevice));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        unsigned long long* activity_output;
        GPUNTT_CUDA_CHECK(hipMalloc(&activity_output,
                                     64 * 512 * sizeof(unsigned long long)));
        GPU_ACTIVITY_HOST(activity_output, 111111);
        GPUNTT_CUDA_CHECK(hipFree(activity_output));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        Modulus64* modulus_device;
        GPUNTT_CUDA_CHECK(hipMalloc(&modulus_device, sizeof(Modulus64)));

        Modulus64 test_modulus_[1] = {modulus};

        GPUNTT_CUDA_CHECK(hipMemcpy(modulus_device, test_modulus_,
                                     sizeof(Modulus64),
                                     hipMemcpyHostToDevice));

        Ninverse64* ninverse_device;
        GPUNTT_CUDA_CHECK(hipMalloc(&ninverse_device, sizeof(Ninverse64)));

        Ninverse64 test_ninverse_[1] = {n_inv};

        GPUNTT_CUDA_CHECK(hipMemcpy(ninverse_device, test_ninverse_,
                                     sizeof(Ninverse64),
                                     hipMemcpyHostToDevice));

        ntt4step_configuration<Data64> cfg_ntt = {
            .n_power = LOGN, .ntt_type = FORWARD, .stream = 0};

        float time = 0;
        hipEvent_t startx, stopx;
        hipEventCreate(&startx);
        hipEventCreate(&stopx);

        hipEventRecord(startx);

        // GPU_4STEP_NTT(Input_Datas, Output_Datas, Forward_Omega_Table1_Device,
        //               Forward_Omega_Table2_Device, W_Table_Device,
        //               modulus_device, cfg_ntt, BATCH, 1);

        GPU_4STEP_NTT(Input_Datas, Output_Datas, Forward_Omega_Table1_Device,
                      Forward_Omega_Table2_Device, W_Table_Device, modulus,
                      cfg_ntt, BATCH);

        hipEventRecord(stopx);
        hipEventSynchronize(stopx);
        hipEventElapsedTime(&time, startx, stopx);
        time_measurements[loop] = time;

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        GPUNTT_CUDA_CHECK(hipFree(Input_Datas));
        GPUNTT_CUDA_CHECK(hipFree(Output_Datas));
        GPUNTT_CUDA_CHECK(hipFree(Forward_Omega_Table1_Device));
        GPUNTT_CUDA_CHECK(hipFree(Forward_Omega_Table2_Device));
        GPUNTT_CUDA_CHECK(hipFree(W_Table_Device));
    }

    cout << endl
         << endl
         << "Average: " << calculate_mean(time_measurements, test_count)
         << endl;
    cout << "Best Average: "
         << find_min_average(time_measurements, test_count, bestof) << endl;

    cout << "Standart Deviation: "
         << calculate_standard_deviation(time_measurements, test_count) << endl;

    return EXIT_SUCCESS;
}