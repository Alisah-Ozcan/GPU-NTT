#include <cstdlib>
#include <random>

#include "ntt_4step.cuh"
#include "ntt_4step_cpu.cuh"

#define DEFAULT_MODULUS

using namespace std;

int LOGN;
int BATCH;

int main(int argc, char* argv[])
{
    CudaDevice();

    if(argc < 3)
    {
        LOGN = 12;
        BATCH = 1;
    }
    else
    {
        LOGN = atoi(argv[1]);
        BATCH = atoi(argv[2]);
        
        if((LOGN < 12) || (24 < LOGN))
        {
            throw std::runtime_error("LOGN should be in range 12 to 24.");
        }
    }

#ifdef BARRETT_64
    ModularReductionType modular_reduction_type = ModularReductionType::BARRET;
#elif defined(GOLDILOCKS_64)
    ModularReductionType modular_reduction_type = ModularReductionType::GOLDILOCK;
#elif defined(PLANTARD_64)
    ModularReductionType modular_reduction_type = ModularReductionType::PLANTARD;
#else
#error "Please define reduction type."
#endif

    // Current 4step NTT implementation only works for ReductionPolynomial::X_N_minus!
    NTTParameters4Step parameters(LOGN, modular_reduction_type, ReductionPolynomial::X_N_minus);

    int N = parameters.n;

    const int test_count = 100;
    const int bestof = 25;
    float time_measurements[test_count];
    for(int loop = 0; loop < test_count; loop++)
    {
        std::random_device rd;
        std::mt19937 gen(rd());
        unsigned long long minNumber = (unsigned long long)1 << 40;
        unsigned long long maxNumber = ((unsigned long long)1 << 40) - 1;
        std::uniform_int_distribution<unsigned long long> dis(minNumber, maxNumber);
        unsigned long long number = dis(gen);

        std::uniform_int_distribution<unsigned long long> dis2(0, number);

        Modulus modulus(number);

        // Random data generation for polynomials
        vector<vector<Data>> input1(BATCH);
        for(int j = 0; j < BATCH; j++)
        {
            for(int i = 0; i < N; i++)
            {
                input1[j].push_back(dis2(gen));
            }
        }

        vector<Root_> forward_root_table1;
#ifdef PLANTARD_64
        for(int i = 0; i<(parameters.n1 >> 1); i++)
        {
            __uint128_t forward =
                ((__uint128_t)(dis(gen)) << (__uint128_t)64) + ((__uint128_t)(dis(gen)));
            forward_root_table1.push_back(forward);
        }
        Ninverse n_inv = {.x = dis(gen), .y = dis(gen)};
#else
        for(int i = 0; i<(parameters.n1 >> 1); i++)
        {
            forward_root_table1.push_back(dis2(gen));
        }
        Ninverse n_inv = dis2(gen);
#endif

        vector<Root_> forward_root_table2;
#ifdef PLANTARD_64
        for(int i = 0; i<(parameters.n2 >> 1); i++)
        {
            __uint128_t forward =
                ((__uint128_t)(dis(gen)) << (__uint128_t)64) + ((__uint128_t)(dis(gen)));
            forward_root_table2.push_back(forward);
        }
#else
        for(int i = 0; i<(parameters.n2 >> 1); i++)
        {
            forward_root_table2.push_back(dis2(gen));
        }
#endif

        vector<Root_> W_root_table;
#ifdef PLANTARD_64
        for(int i = 0; i<parameters.n; i++)
        {
            __uint128_t forward =
                ((__uint128_t)(dis(gen)) << (__uint128_t)64) + ((__uint128_t)(dis(gen)));
            W_root_table.push_back(forward);
        }
#else
        for(int i = 0; i<parameters.n; i++)
        {
            W_root_table.push_back(dis2(gen));
        }
#endif

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        Data* Input_Datas;

        THROW_IF_CUDA_ERROR(hipMalloc(&Input_Datas, BATCH * N * sizeof(Data)));

        for(int j = 0; j < BATCH; j++)
        {
            THROW_IF_CUDA_ERROR(hipMemcpy(Input_Datas + (N * j), input1[j].data(),
                                           N * sizeof(Data), hipMemcpyHostToDevice));
        }

        Data* Output_Datas;

        THROW_IF_CUDA_ERROR(hipMalloc(&Output_Datas, BATCH * N * sizeof(Data)));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        Root* Forward_Omega_Table1_Device;
        THROW_IF_CUDA_ERROR(
            hipMalloc(&Forward_Omega_Table1_Device, (parameters.n1 >> 1) * sizeof(Root)));
        THROW_IF_CUDA_ERROR(hipMemcpy(Forward_Omega_Table1_Device, forward_root_table1.data(),
                                       (parameters.n1 >> 1) * sizeof(Root),
                                       hipMemcpyHostToDevice));

        Root* Forward_Omega_Table2_Device;
        THROW_IF_CUDA_ERROR(
            hipMalloc(&Forward_Omega_Table2_Device, (parameters.n2 >> 1) * sizeof(Root)));
        THROW_IF_CUDA_ERROR(hipMemcpy(Forward_Omega_Table2_Device, forward_root_table2.data(),
                                       (parameters.n2 >> 1) * sizeof(Root),
                                       hipMemcpyHostToDevice));

        Root* W_Table_Device;
        THROW_IF_CUDA_ERROR(hipMalloc(&W_Table_Device, parameters.n * sizeof(Root)));
        THROW_IF_CUDA_ERROR(hipMemcpy(W_Table_Device, W_root_table.data(),
                                       (parameters.n >> 1) * sizeof(Root), hipMemcpyHostToDevice));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        unsigned long long* activity_output;
        THROW_IF_CUDA_ERROR(hipMalloc(&activity_output, 64 * 512 * sizeof(unsigned long long)));
        GPU_ACTIVITY_HOST(activity_output, 111111);
        THROW_IF_CUDA_ERROR(hipFree(activity_output));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        Modulus* modulus_device;
        THROW_IF_CUDA_ERROR(hipMalloc(&modulus_device, sizeof(Modulus)));

        Modulus test_modulus_[1] = {modulus};

        THROW_IF_CUDA_ERROR(
            hipMemcpy(modulus_device, test_modulus_, sizeof(Modulus), hipMemcpyHostToDevice));

        Ninverse* ninverse_device;
        THROW_IF_CUDA_ERROR(hipMalloc(&ninverse_device, sizeof(Ninverse)));

        Ninverse test_ninverse_[1] = {n_inv};

        THROW_IF_CUDA_ERROR(
            hipMemcpy(ninverse_device, test_ninverse_, sizeof(Ninverse), hipMemcpyHostToDevice));

        ntt4step_configuration cfg_ntt = {.n_power = LOGN,
                                         .ntt_type = FORWARD,
                                         .stream = 0};

        float time = 0;
        hipEvent_t startx, stopx;
        hipEventCreate(&startx);
        hipEventCreate(&stopx);

        hipEventRecord(startx);

        //GPU_4STEP_NTT(Input_Datas, Output_Datas, Forward_Omega_Table1_Device,
        //              Forward_Omega_Table2_Device, W_Table_Device, modulus_device, cfg_ntt, BATCH,
        //              1);

        GPU_4STEP_NTT(Input_Datas, Output_Datas, Forward_Omega_Table1_Device,
                      Forward_Omega_Table2_Device, W_Table_Device, modulus, cfg_ntt, BATCH);

        hipEventRecord(stopx);
        hipEventSynchronize(stopx);
        hipEventElapsedTime(&time, startx, stopx);
        time_measurements[loop] = time;

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        THROW_IF_CUDA_ERROR(hipFree(Input_Datas));
        THROW_IF_CUDA_ERROR(hipFree(Output_Datas));
        THROW_IF_CUDA_ERROR(hipFree(Forward_Omega_Table1_Device));
        THROW_IF_CUDA_ERROR(hipFree(Forward_Omega_Table2_Device));
        THROW_IF_CUDA_ERROR(hipFree(W_Table_Device));
    }

    cout << endl << endl << "Average: " << calculate_mean(time_measurements, test_count) << endl;
    cout << "Best Average: " << find_min_average(time_measurements, test_count, bestof) << endl;

    cout << "Standart Deviation: " << calculate_standard_deviation(time_measurements, test_count)
         << endl;

    return EXIT_SUCCESS;
}