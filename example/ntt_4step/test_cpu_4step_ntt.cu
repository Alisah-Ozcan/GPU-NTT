// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include <cstdlib> // For atoi or atof functions
#include <fstream>
#include <random>

#include "ntt.cuh"
#include "ntt_4step_cpu.cuh"

#define DEFAULT_MODULUS

using namespace std;
using namespace gpuntt;

int LOGN;
int BATCH;
int N;

//typedef Data32 TestDataType; // Use for 32-bit Test
typedef Data64 TestDataType; // Use for 64-bit Test

int main(int argc, char* argv[])
{
    CudaDevice();

    if (argc < 3)
    {
        LOGN = 12;
        BATCH = 1;
    }
    else
    {
        LOGN = atoi(argv[1]);
        BATCH = atoi(argv[2]);
    }

    NTTParameters4Step<TestDataType> parameters(LOGN, ReductionPolynomial::X_N_minus);

    // NTT generator with certain modulus and root of unity
    NTT_4STEP_CPU<TestDataType> generator(parameters);

    std::random_device rd;
    std::mt19937 gen(rd());
    unsigned long long minNumber = 0;
    unsigned long long maxNumber = parameters.modulus.value - 1;
    std::uniform_int_distribution<unsigned long long> dis(minNumber, maxNumber);

    // Random data generation for polynomials
    vector<TestDataType> input1;
    vector<TestDataType> input2;
    for (int j = 0; j < BATCH; j++)
    {
        for (int i = 0; i < parameters.n; i++)
        {
            input1.push_back(dis(gen));
            input2.push_back(dis(gen));
        }
    }

    // Performing CPU NTT
    vector<TestDataType> ntt_input1 = generator.ntt(input1);
    vector<TestDataType> ntt_input2 = generator.ntt(input2);
    vector<TestDataType> output = generator.mult(ntt_input1, ntt_input2);
    vector<TestDataType> ntt_mult_result = generator.intt(output);

    // Comparing CPU NTT multiplication results and schoolbook multiplication
    // results
    bool check = true;
    std::vector<TestDataType> schoolbook_result = schoolbook_poly_multiplication(
        input1, input2, parameters.modulus, ReductionPolynomial::X_N_minus);

    check = check_result(ntt_mult_result.data(), schoolbook_result.data(),
                         parameters.n);

    if (check)
    {
        cout << "All Correct." << endl;
    }

    return EXIT_SUCCESS;
}
