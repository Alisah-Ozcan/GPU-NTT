// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include <cstdlib>
#include <random>

#include "ntt.cuh"
#include "ntt_4step.cuh"
#include "ntt_4step_cpu.cuh"

#define DEFAULT_MODULUS

using namespace std;
using namespace gpuntt;

int LOGN;
int BATCH;
int N;

// typedef Data32 TestDataType; // Use for 32-bit Test
typedef Data64 TestDataType; // Use for 64-bit Test

int main(int argc, char* argv[])
{
    CudaDevice();

    if (argc < 3)
    {
        LOGN = 12;
        BATCH = 1;
    }
    else
    {
        LOGN = atoi(argv[1]);
        BATCH = atoi(argv[2]);
    }

    // Current 4step NTT implementation only works for
    // ReductionPolynomial::X_N_minus!
    NTTParameters4Step<TestDataType> parameters(LOGN,
                                                ReductionPolynomial::X_N_minus);

    // NTT generator with certain modulus and root of unity
    NTT_4STEP_CPU<TestDataType> generator(parameters);

    std::random_device rd;
    std::mt19937 gen(rd());
    unsigned long long minNumber = 0;
    unsigned long long maxNumber = parameters.modulus.value - 1;
    std::uniform_int_distribution<unsigned long long> dis(minNumber, maxNumber);

    // Random data generation for polynomials
    vector<vector<TestDataType>> input1(BATCH);
    for (int j = 0; j < BATCH; j++)
    {
        for (int i = 0; i < parameters.n; i++)
        {
            input1[j].push_back(dis(gen));
        }
    }

    // Performing CPU NTT
    vector<vector<TestDataType>> ntt_result(BATCH);
    for (int i = 0; i < BATCH; i++)
    {
        ntt_result[i] = generator.ntt(input1[i]);
    }
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////

    TestDataType* Input_Datas;

    GPUNTT_CUDA_CHECK(
        hipMalloc(&Input_Datas, BATCH * parameters.n * sizeof(TestDataType)));

    TestDataType* Output_Datas;
    GPUNTT_CUDA_CHECK(
        hipMalloc(&Output_Datas, BATCH * parameters.n * sizeof(TestDataType)));

    for (int j = 0; j < BATCH; j++)
    {
        GPUNTT_CUDA_CHECK(hipMemcpy(
            Input_Datas + (parameters.n * j), input1[j].data(),
            parameters.n * sizeof(TestDataType), hipMemcpyHostToDevice));
    }

    //////////////////////////////////////////////////////////////////////////

    vector<Root<TestDataType>> psitable1 =
        parameters.gpu_root_of_unity_table_generator(
            parameters.n1_based_root_of_unity_table);
    Root<TestDataType>* psitable_device1;
    GPUNTT_CUDA_CHECK(hipMalloc(
        &psitable_device1, (parameters.n1 >> 1) * sizeof(Root<TestDataType>)));
    GPUNTT_CUDA_CHECK(
        hipMemcpy(psitable_device1, psitable1.data(),
                   (parameters.n1 >> 1) * sizeof(Root<TestDataType>),
                   hipMemcpyHostToDevice));

    vector<Root<TestDataType>> psitable2 =
        parameters.gpu_root_of_unity_table_generator(
            parameters.n2_based_root_of_unity_table);
    Root<TestDataType>* psitable_device2;
    GPUNTT_CUDA_CHECK(hipMalloc(
        &psitable_device2, (parameters.n2 >> 1) * sizeof(Root<TestDataType>)));
    GPUNTT_CUDA_CHECK(
        hipMemcpy(psitable_device2, psitable2.data(),
                   (parameters.n2 >> 1) * sizeof(Root<TestDataType>),
                   hipMemcpyHostToDevice));

    Root<TestDataType>* W_Table_device;
    GPUNTT_CUDA_CHECK(
        hipMalloc(&W_Table_device, parameters.n * sizeof(Root<TestDataType>)));
    GPUNTT_CUDA_CHECK(hipMemcpy(
        W_Table_device, parameters.W_root_of_unity_table.data(),
        parameters.n * sizeof(Root<TestDataType>), hipMemcpyHostToDevice));

    //////////////////////////////////////////////////////////////////////////

    Modulus<TestDataType>* test_modulus;
    GPUNTT_CUDA_CHECK(hipMalloc(&test_modulus, sizeof(Modulus<TestDataType>)));

    Modulus<TestDataType> test_modulus_[1] = {parameters.modulus};

    GPUNTT_CUDA_CHECK(hipMemcpy(test_modulus, test_modulus_,
                                 sizeof(Modulus<TestDataType>),
                                 hipMemcpyHostToDevice));

    Ninverse<TestDataType>* test_ninverse;
    GPUNTT_CUDA_CHECK(
        hipMalloc(&test_ninverse, sizeof(Ninverse<TestDataType>)));

    Ninverse<TestDataType> test_ninverse_[1] = {parameters.n_inv};

    GPUNTT_CUDA_CHECK(hipMemcpy(test_ninverse, test_ninverse_,
                                 sizeof(Ninverse<TestDataType>),
                                 hipMemcpyHostToDevice));

    ntt4step_rns_configuration<TestDataType> cfg_intt = {.n_power = LOGN,
                                                         .ntt_type = FORWARD,
                                                         .mod_inverse =
                                                             test_ninverse,
                                                         .stream = 0};

    //////////////////////////////////////////////////////////////////////////
    GPU_Transpose(Input_Datas, Output_Datas, parameters.n1, parameters.n2,
                  parameters.logn, BATCH);

    GPU_4STEP_NTT(Output_Datas, Input_Datas, psitable_device1, psitable_device2,
                  W_Table_device, test_modulus, cfg_intt, BATCH, 1);

    GPU_Transpose(Input_Datas, Output_Datas, parameters.n1, parameters.n2,
                  parameters.logn, BATCH);

    vector<TestDataType> Output_Host(parameters.n * BATCH);
    hipMemcpy(Output_Host.data(), Output_Datas,
               parameters.n * BATCH * sizeof(TestDataType),
               hipMemcpyDeviceToHost);

    // Comparing GPU NTT results and CPU NTT results
    bool check = true;
    for (int i = 0; i < BATCH; i++)
    {
        check = check_result(Output_Host.data() + (i * parameters.n),
                             ntt_result[i].data(), parameters.n);

        if (!check)
        {
            cout << "(in " << i << ". Poly.)" << endl;
            break;
        }

        if ((i == (BATCH - 1)) && check)
        {
            cout << "All Correct." << endl;
        }
    }

    return EXIT_SUCCESS;
}