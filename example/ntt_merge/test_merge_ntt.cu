// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include <cstdlib>
#include <random>

#include "ntt.cuh"

#define DEFAULT_MODULUS

using namespace std;
using namespace gpuntt;

int LOGN;
int BATCH;

int main(int argc, char* argv[])
{
    CudaDevice();

    int device = 0; // Assuming you are using device 0
    hipSetDevice(device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    std::cout << "Maximum Grid Size: " << prop.maxGridSize[0] << " x "
              << prop.maxGridSize[1] << " x " << prop.maxGridSize[2]
              << std::endl;

    if (argc < 3)
    {
        LOGN = 12;
        BATCH = 1;
    }
    else
    {
        LOGN = atoi(argv[1]);
        BATCH = atoi(argv[2]);
    }

    ModularReductionType modular_reduction_type = ModularReductionType::BARRET;

#ifdef DEFAULT_MODULUS
    NTTParameters<Data64> parameters(LOGN, modular_reduction_type,
                                     ReductionPolynomial::X_N_minus);
#else
    NTTFactors factor((Modulus) 576460752303415297, 288482366111684746,
                      238394956950829);
    NTTParameters parameters(LOGN, factor, ReductionPolynomial::X_N_minus);
#endif

    // NTT generator with certain modulus and root of unity
    NTTCPU<Data64> generator(parameters);

    std::random_device rd;
    // std::mt19937 gen(rd());
    std::mt19937 gen(0);
    unsigned long long minNumber = 0;
    unsigned long long maxNumber = parameters.modulus.value - 1;
    std::uniform_int_distribution<unsigned long long> dis(minNumber, maxNumber);

    // Random data generation for polynomials
    vector<vector<Data64>> input1(BATCH);
    for (int j = 0; j < BATCH; j++)
    {
        for (int i = 0; i < parameters.n; i++)
        {
            input1[j].push_back(dis(gen));
        }
    }

    // Performing CPU NTT
    vector<vector<Data64>> ntt_result(BATCH);
    for (int i = 0; i < BATCH; i++)
    {
        ntt_result[i] = generator.ntt(input1[i]);
    }
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////

    Data64* InOut_Datas;

    GPUNTT_CUDA_CHECK(
        hipMalloc(&InOut_Datas, BATCH * parameters.n * sizeof(Data64)));

    for (int j = 0; j < BATCH; j++)
    {
        GPUNTT_CUDA_CHECK(
            hipMemcpy(InOut_Datas + (parameters.n * j), input1[j].data(),
                       parameters.n * sizeof(Data64), hipMemcpyHostToDevice));
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////

    Root64* Forward_Omega_Table_Device;

    GPUNTT_CUDA_CHECK(
        hipMalloc(&Forward_Omega_Table_Device,
                   parameters.root_of_unity_size * sizeof(Root64)));

    vector<Root64> forward_omega_table =
        parameters.gpu_root_of_unity_table_generator(
            parameters.forward_root_of_unity_table);

    GPUNTT_CUDA_CHECK(hipMemcpy(Forward_Omega_Table_Device,
                                 forward_omega_table.data(),
                                 parameters.root_of_unity_size * sizeof(Root64),
                                 hipMemcpyHostToDevice));

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////

    Modulus64* test_modulus;
    GPUNTT_CUDA_CHECK(hipMalloc(&test_modulus, sizeof(Modulus64)));

    Modulus64 test_modulus_[1] = {parameters.modulus};

    GPUNTT_CUDA_CHECK(hipMemcpy(test_modulus, test_modulus_, sizeof(Modulus64),
                                 hipMemcpyHostToDevice));

    ntt_rns_configuration<Data64> cfg_ntt = {.n_power = LOGN,
                                             .ntt_type = FORWARD,
                                             .reduction_poly =
                                                 ReductionPolynomial::X_N_minus,
                                             .zero_padding = false,
                                             .stream = 0};
    GPU_NTT_Inplace(InOut_Datas, Forward_Omega_Table_Device, test_modulus,
                    cfg_ntt, BATCH, 1);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////

    Data64* Output_Host;

    Output_Host = (Data64*) malloc(BATCH * parameters.n * sizeof(Data64));

    GPUNTT_CUDA_CHECK(hipMemcpy(Output_Host, InOut_Datas,
                                 BATCH * parameters.n * sizeof(Data64),
                                 hipMemcpyDeviceToHost));

    // Comparing GPU NTT results and CPU NTT results
    bool check = true;
    for (int i = 0; i < BATCH; i++)
    {
        check = check_result(Output_Host + (i * parameters.n),
                             ntt_result[i].data(), parameters.n);

        if (!check)
        {
            cout << "(in " << i << ". Poly.)" << endl;
            break;
        }

        if ((i == (BATCH - 1)) && check)
        {
            cout << "All Correct." << endl;
        }
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////

    GPUNTT_CUDA_CHECK(hipFree(InOut_Datas));
    GPUNTT_CUDA_CHECK(hipFree(Forward_Omega_Table_Device));
    free(Output_Host);

    return EXIT_SUCCESS;
}