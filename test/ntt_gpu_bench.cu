#include <cstdlib>  // For atoi or atof functions
#include <random>

#include "../src/ntt.cuh"

#define DEFAULT_MODULUS

using namespace std;

int LOGN;
int BATCH;

int main(int argc, char* argv[])
{
    CudaDevice();

    if (argc < 3)
    {
        LOGN = 12;
        BATCH = 1;
    }
    else
    {
        LOGN = atoi(argv[1]);
        BATCH = atoi(argv[2]);
    }

    // NTT generator with certain modulus and root of unity

    int N = 1 << LOGN;
    int ROOT_SIZE = N >> 1;

    const int test_count = 100;
    const int bestof = 25;
    float time_measurements[test_count];
    for (int loop = 0; loop < test_count; loop++)
    {
        std::random_device rd;
        std::mt19937 gen(rd());
        unsigned long long minNumber = (unsigned long long)1 << 40;
        unsigned long long maxNumber = ((unsigned long long)1 << 40) - 1;
        std::uniform_int_distribution<unsigned long long> dis(minNumber,
                                                              maxNumber);
        unsigned long long number = dis(gen);

        std::uniform_int_distribution<unsigned long long> dis2(0, number);

        Modulus modulus(number);

        // Random data generation for polynomials
        vector<vector<Data>> input1(BATCH);
        for (int j = 0; j < BATCH; j++)
        {
            for (int i = 0; i < N; i++)
            {
                input1[j].push_back(dis2(gen));
            }
        }

        vector<Root_> forward_root_table;
        vector<Root_> inverse_root_table;
#ifdef PLANTARD_64
        for (int i = 0; i < ROOT_SIZE; i++)
        {
            __uint128_t forward = ((__uint128_t)(dis(gen)) << (__uint128_t)64) +
                                  ((__uint128_t)(dis(gen)));
            __uint128_t inverse = ((__uint128_t)(dis(gen)) << (__uint128_t)64) +
                                  ((__uint128_t)(dis(gen)));
            forward_root_table.push_back(forward);
            inverse_root_table.push_back(inverse);
        }

        Ninverse n_inv = {.x = dis(gen), .y = dis(gen)};
#else
        for (int i = 0; i < ROOT_SIZE; i++)
        {
            forward_root_table.push_back(dis2(gen));
            inverse_root_table.push_back(dis2(gen));
        }
        Ninverse n_inv = dis2(gen);
#endif

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        Data* InOut_Datas;

        THROW_IF_CUDA_ERROR(hipMalloc(&InOut_Datas, BATCH * N * sizeof(Data)));

        for (int j = 0; j < BATCH; j++)
        {
            THROW_IF_CUDA_ERROR(hipMemcpy(InOut_Datas + (N * j),
                                           input1[j].data(), N * sizeof(Data),
                                           hipMemcpyHostToDevice));
        }

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        Root* Forward_Omega_Table_Device;

        THROW_IF_CUDA_ERROR(
            hipMalloc(&Forward_Omega_Table_Device, ROOT_SIZE * sizeof(Root)));

        THROW_IF_CUDA_ERROR(
            hipMemcpy(Forward_Omega_Table_Device, forward_root_table.data(),
                       ROOT_SIZE * sizeof(Root), hipMemcpyHostToDevice));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        Root* Inverse_Omega_Table_Device;

        THROW_IF_CUDA_ERROR(
            hipMalloc(&Inverse_Omega_Table_Device, ROOT_SIZE * sizeof(Root)));

        THROW_IF_CUDA_ERROR(
            hipMemcpy(Inverse_Omega_Table_Device, inverse_root_table.data(),
                       ROOT_SIZE * sizeof(Root), hipMemcpyHostToDevice));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        unsigned long long* activity_output;
        THROW_IF_CUDA_ERROR(hipMalloc(&activity_output,
                                       64 * 512 * sizeof(unsigned long long)));
        GPU_ACTIVITY_HOST(activity_output, 111111);
        THROW_IF_CUDA_ERROR(hipFree(activity_output));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        ntt_configuration cfg_ntt = {
            .n_power = LOGN,
            .ntt_type = FORWARD,
            .reduction_poly = ReductionPolynomial::X_N_minus,
            .zero_padding = false,
            .stream = 0};

        ntt_configuration cfg_intt = {
            .n_power = LOGN,
            .ntt_type = INVERSE,
            .reduction_poly = ReductionPolynomial::X_N_minus,
            .zero_padding = false,
            .mod_inverse = n_inv,
            .stream = 0};

        float time = 0;
        hipEvent_t startx, stopx;
        hipEventCreate(&startx);
        hipEventCreate(&stopx);

        hipEventRecord(startx);
        GPU_NTT(InOut_Datas, Forward_Omega_Table_Device, modulus, cfg_ntt,
                BATCH);

        hipEventRecord(stopx);
        hipEventSynchronize(stopx);
        hipEventElapsedTime(&time, startx, stopx);
        time_measurements[loop] = time;

        // GPU_NTT(InOut_Datas, Inverse_Omega_Table_Device, modulus,
        //         cfg_intt, BATCH);

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////

        THROW_IF_CUDA_ERROR(hipFree(InOut_Datas));
        THROW_IF_CUDA_ERROR(hipFree(Forward_Omega_Table_Device));
        THROW_IF_CUDA_ERROR(hipFree(Inverse_Omega_Table_Device));
    }

    cout << endl
         << endl
         << "Average: " << calculate_mean(time_measurements, test_count)
         << endl;
    cout << "Best Average: "
         << find_min_average(time_measurements, test_count, bestof) << endl;

    cout << "Standart Deviation: "
         << calculate_standard_deviation(time_measurements, test_count) << endl;

    return EXIT_SUCCESS;
}

